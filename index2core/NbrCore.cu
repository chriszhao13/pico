#include "hip/hip_runtime.h"
//
// Created by Chris on 2022/11/30.
//
#include "shared/graph.h"
#include "shared/Timer.h"
#include "shared/util.h"
#include "shared/virtual_graph.h"
#include "shared/globals.h"
#include "shared/argument.h"
#include "shared/gpu_error.cuh"

__global__ void core_kernel(

    uint partSize,

    unsigned int numParts,

    unsigned int *nodePointer,

    PartPointer *partNodePointer,

    const unsigned int *edgeList,

    unsigned int *histo,

    unsigned int *core,

    bool *sighed)
{

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts)
    {

        unsigned int id = partNodePointer[partId].node;

        unsigned int part = partNodePointer[partId].part;

        if (!sighed[id])
            return;

        unsigned int thisPointer = nodePointer[id];

        unsigned int degree = edgeList[thisPointer];

        int numParts;

        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int nbr;

        unsigned int ofs = thisPointer + part + 1;

        //
        histo[thisPointer] = degree;

        for (int i = 0; i < partSize; i++)
        {

            if (part + i * numParts >= degree)
                break;

            nbr = ofs + i * numParts;
            unsigned int ofs_nbr = min(degree, core[edgeList[nbr]]);

            atomicAdd(histo + thisPointer + ofs_nbr, 1);
        }
    }
}

__global__ void count_kernel(
    unsigned int num_node,
    unsigned int *nodePointer,
    unsigned int *edgeList,
    unsigned int *histo,
    unsigned int *core,
    bool *finished,
    bool *sighed,
    bool *nbr_sighed)
{
    unsigned int nodeId = blockDim.x * blockIdx.x + threadIdx.x;
    if (!sighed[nodeId] || nodeId >= num_node)
        return;
    unsigned int thisPointer = nodePointer[nodeId];
    unsigned int node_old_core = core[nodeId];
    unsigned int sum = 0;
    unsigned int k;
    unsigned int degree = edgeList[thisPointer];

    for (k = degree; k >= 1; k--)
    {

        sum += histo[thisPointer + k];

        if (sum >= k)
            break;
    }

    if (k == node_old_core)
        return;

    *finished = false;
    histo[thisPointer + k] = sum;
    core[nodeId] = k;
    nbr_sighed[nodeId] = true;
}

__global__ void nbr_kernel(

    uint partSize,

    unsigned int numParts,
    unsigned int *nodePointer,
    PartPointer *partNodePointer,
    unsigned int *edgeList,
    bool *sighed,
    bool *nbr_sighed)
{

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts)
    {
        unsigned int id = partNodePointer[partId].node;
        unsigned int part = partNodePointer[partId].part;

        if (!nbr_sighed[id])
        {
            return;
        }

        unsigned int thisPointer = nodePointer[id];
        unsigned int degree = edgeList[thisPointer];

        int numParts;

        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int nbr;

        unsigned int ofs = thisPointer + part + 1;

        for (int i = 0; i < partSize; i++)
        {

            if (part + i * numParts >= degree)
                break;

            nbr = ofs + i * numParts;

            sighed[edgeList[nbr]] = true;
        }
    }
}

int main(int argc, char **argv)
{

    argument arguments(argc, argv, false, false);

    Graph graph(arguments.input, false);

    graph.ReadGraph();

    VirtualGraph vGraph(graph, arguments.Part_Size);

    vGraph.MakeUGraph();

    uint Part_Size = arguments.Part_Size;

    uint num_nodes = graph.num_nodes;
    uint num_edges = 2 * graph.num_edges;

  
    hipSetDevice(arguments.deviceID);

    hipFree(0);

    uint *core = new uint[num_nodes];

    for (int i = 0; i < num_nodes; i++)
    {
        core[i] = vGraph.outDegree[i];
    }

    unsigned int *d_nodePointer;
    unsigned int *d_edgeList;
    unsigned int *d_core;
    unsigned int *d_degree;
    unsigned int *d_histo;
    PartPointer *d_partNodePointer;

    bool finished = true;
    bool *d_finished;

    bool *d_sighed;
    bool *d_nbr_sighed;

    gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_edgeList, (num_edges + num_nodes) * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_core, num_nodes * sizeof(unsigned int)));

    gpuErrorcheck(hipMalloc(&d_degree, num_nodes * sizeof(unsigned int)));

    gpuErrorcheck(hipMalloc(&d_histo, (num_edges + num_nodes) * sizeof(unsigned int)));

    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));

    gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));
    gpuErrorcheck(hipMalloc(&d_sighed, num_nodes * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_nbr_sighed, num_nodes * sizeof(bool)));
    gpuErrorcheck(
        hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (num_edges + num_nodes) * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_core, core, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_degree, core, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer),
                             hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemset(d_sighed, true, num_nodes * sizeof(bool)));
    gpuErrorcheck(hipMemset(d_nbr_sighed, false, num_nodes * sizeof(bool)));

    int itr = 0;
    Timer t{};
    t.Start();

    while (true)
    {

        itr++;

        finished = true;
        hipMemset(d_histo, 0, (num_edges + num_nodes) * sizeof(unsigned int));
        hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice);

        core_kernel<<<vGraph.numParts / 512 + 1, 512>>>(
            Part_Size,
            vGraph.numParts,
            d_nodePointer,
            d_partNodePointer,
            d_edgeList,
            d_histo,

            d_core,

            d_sighed);

        count_kernel<<<num_nodes / 512 + 1, 512>>>(
            num_nodes,
            d_nodePointer,
            d_edgeList,
            d_histo,
            d_core,
            d_finished,
            d_sighed,
            d_nbr_sighed);

        hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);

        hipMemset(d_sighed, false, num_nodes * sizeof(bool));

        if (finished)
            break;

        nbr_kernel<<<vGraph.numParts / 512 + 1, 512>>>(
            Part_Size,

            vGraph.numParts,
    
            d_nodePointer,
     
            d_partNodePointer,

            d_edgeList,

            d_sighed,

            d_nbr_sighed);
        hipMemset(d_nbr_sighed, false, num_nodes * sizeof(bool));
    }
    //

    double runtime = t.Finish();
    //
    cout << "Number of iterations = " << itr << endl;

    cout << "Processing finished in " << runtime << " (ms).\n";

    gpuErrorcheck(hipMemcpy(core, d_core, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
    utility::PrintMaxMin(core, num_nodes);

    gpuErrorcheck(hipFree(d_nodePointer));
    gpuErrorcheck(hipFree(d_edgeList));
    gpuErrorcheck(hipFree(d_core));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_partNodePointer));
    gpuErrorcheck(hipFree(d_sighed));
    gpuErrorcheck(hipFree(d_nbr_sighed));
    gpuErrorcheck(hipFree(d_histo))
    gpuErrorcheck(hipFree(d_degree))

            return 0;
}
