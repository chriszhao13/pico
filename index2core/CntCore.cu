#include "hip/hip_runtime.h"
//
// Created by Chris on 2022/11/30.
//
#include "shared/graph.h"
#include "shared/Timer.h"
#include "shared/util.h"
#include "shared/virtual_graph.h"
#include "shared/globals.h"
#include "shared/argument.h"
#include "shared/gpu_error.cuh"

#define printMinMaxOneStep 1

__device__ int DEVICE_count_change = 0;
__device__ int DEVICE_nbr = 0;

__global__ void core_kernel(

    uint partSize,

    unsigned int numParts,

    unsigned int *nodePointer,

    PartPointer *partNodePointer,

    const unsigned int *edgeList,

    unsigned int *histo,

    unsigned int *core,

    unsigned int *cnt)
{

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts)
    {

        unsigned int id = partNodePointer[partId].node;

        unsigned int part = partNodePointer[partId].part;

        if (cnt[id] >= core[id])
            return;

        unsigned int thisPointer = nodePointer[id];

        unsigned int degree = edgeList[thisPointer];

        int numParts;

        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int nbr;

        unsigned int ofs = thisPointer + part + 1;
        //

        for (int i = 0; i < partSize; i++)
        {

            if (part + i * numParts >= degree)
                break;

            nbr = ofs + i * numParts;

            unsigned int ofs_nbr = min(degree, core[edgeList[nbr]]);

            atomicAdd(histo + thisPointer + ofs_nbr, 1);
        }
    }
}

__global__ void count_kernel(

    unsigned int num_node,

    unsigned int *nodePointer,

    unsigned int *histo,

    unsigned int *core,

    const unsigned int *edgeList,

    unsigned int *cnt,

    bool *finished,

    bool *sighed)
{

    unsigned int nodeId = blockDim.x * blockIdx.x + threadIdx.x;

    if (cnt[nodeId] >= core[nodeId] || nodeId >= num_node)
        return;

    unsigned int thisPointer = nodePointer[nodeId];

    unsigned int node_old_core = core[nodeId];

    unsigned int sum = 0;

    unsigned int k;

    for (k = edgeList[thisPointer]; k >= 1; k--)
    {

        sum += histo[thisPointer + k];

        if (sum >= k)
            break;
    }
    if (k == core[nodeId])
        return;

    *finished = false;
    histo[thisPointer] = node_old_core;
    core[nodeId] = k;

    sighed[nodeId] = true;
}

__global__ void nbr_kernel(

    uint partSize,

    unsigned int numParts,

    unsigned int *nodePointer,

    PartPointer *partNodePointer,

    unsigned int *edgeList,

    bool *sighed,

    bool *sighedNBR,

    unsigned int *cnt)
{

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts)
    {

        unsigned int id = partNodePointer[partId].node;

        unsigned int part = partNodePointer[partId].part;
        // CSR

        if (!sighed[id])
        {
            return;
        }

        unsigned int thisPointer = nodePointer[id];
        // 前 n 个
        unsigned int degree = edgeList[thisPointer];

        int numParts;

        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int nbr;

        unsigned int ofs = thisPointer + part + 1;

        for (int i = 0; i < partSize; i++)
        {

            if (part + i * numParts >= degree)
                break;

            nbr = ofs + i * numParts;
            cnt[edgeList[nbr]] = 0;
            sighedNBR[edgeList[nbr]] = true;
        }
    }
}

__global__ void cnt_kernel(

    uint partSize,

    unsigned int numParts,

    unsigned int *nodePointer,

    PartPointer *partNodePointer,

    const unsigned int *edgeList,

    unsigned int *histo,

    unsigned int *core,

    unsigned int *cnt,

    bool *sighedNBR)
{

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts)
    {

        unsigned int id = partNodePointer[partId].node;

        unsigned int part = partNodePointer[partId].part;

        if (!sighedNBR[id])
        {
            return;
        }

        unsigned int thisPointer = nodePointer[id];

        unsigned int degree = edgeList[thisPointer];

        int numParts;

        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int nbr;

        unsigned int ofs = thisPointer + part + 1;

        for (int i = 0; i < partSize; i++)
        {

            if (part + i * numParts >= degree)
                break;

            nbr = ofs + i * numParts;

            if (core[edgeList[nbr]] >= core[id])
            {
                atomicAdd(cnt + id, 1);
            }
        }
    }
}

int main(int argc, char **argv)
{

    argument arguments(argc, argv, false, false);

    Graph graph(arguments.input, false);

    graph.ReadGraph();

    VirtualGraph vGraph(graph, arguments.Part_Size);

    vGraph.MakeUGraph();

    uint Part_Size = arguments.Part_Size;

   
    uint num_nodes = graph.num_nodes;
    uint num_edges = 2 * graph.num_edges;

  
    hipSetDevice(arguments.deviceID);

    hipFree(0);

    uint *core = new uint[num_nodes];

    for (int i = 0; i < num_nodes; i++)
    {
        core[i] = vGraph.outDegree[i];
    }
   
    unsigned int *d_nodePointer;
    unsigned int *d_edgeList;
    unsigned int *d_core;
    unsigned int *d_degree;
    unsigned int *d_histo;
    unsigned int *d_cnt;
    PartPointer *d_partNodePointer;

    bool finished = true;
    bool *d_finished;

    bool *d_sighed;

    bool *d_sighedNBR;

    gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_edgeList, (num_edges + num_nodes) * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_core, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_cnt, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_degree, num_nodes * sizeof(unsigned int)));

    gpuErrorcheck(hipMalloc(&d_histo, (num_edges + num_nodes) * sizeof(unsigned int)));

    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));

    gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));
    gpuErrorcheck(hipMalloc(&d_sighed, num_nodes * sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_sighedNBR, num_nodes * sizeof(bool)));
    gpuErrorcheck(
        hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (num_edges + num_nodes) * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_core, core, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_degree, core, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer),
                             hipMemcpyHostToDevice));

    gpuErrorcheck(hipMemset(d_cnt, 0, num_nodes * sizeof(unsigned int)));

   
    int itr = 0;
    Timer t{};
    t.Start();

    while (true)
    {

        itr++;
        finished = true;
        hipMemset(d_sighed, false, num_nodes * sizeof(bool));
        hipMemset(d_sighedNBR, false, num_nodes * sizeof(bool));
        hipMemset(d_histo, 0, (num_edges + num_nodes) * sizeof(unsigned int));
        hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice);

        core_kernel<<<vGraph.numParts / 512 + 1, 512>>>(
            Part_Size,

            vGraph.numParts,

            d_nodePointer,

            d_partNodePointer,

            d_edgeList,

            d_histo,

            d_core,

            d_cnt);

        count_kernel<<<num_nodes / 512 + 1, 512>>>(
            num_nodes,
            d_nodePointer,
            d_histo,
            d_core,
            d_edgeList,
            d_cnt,
            d_finished,
            d_sighed);

        hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);

        if (finished)
            break;

        nbr_kernel<<<vGraph.numParts / 512 + 1, 512>>>(
            Part_Size,

            vGraph.numParts,

            d_nodePointer,

            d_partNodePointer,
            // 边数组
            d_edgeList,

            d_sighed,

            d_sighedNBR,

            d_cnt);

        cnt_kernel<<<vGraph.numParts / 512 + 1, 512>>>(
            Part_Size,

            vGraph.numParts,

            d_nodePointer,

            d_partNodePointer,
            d_edgeList,
            // 边数组
            d_histo,
            d_core,
            d_cnt,
            d_sighedNBR);
    }
    //
    double runtime = t.Finish();
    //
    cout << "Number of iterations = " << itr << endl;

    cout << "Processing finished in " << runtime << " (ms).\n";

    gpuErrorcheck(hipMemcpy(core, d_core, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
    utility::PrintMaxMin(core, num_nodes);

    gpuErrorcheck(hipFree(d_nodePointer));
    gpuErrorcheck(hipFree(d_edgeList));
    gpuErrorcheck(hipFree(d_core));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_partNodePointer));
    gpuErrorcheck(hipFree(d_sighed));
    gpuErrorcheck(hipFree(d_cnt));
    gpuErrorcheck(hipFree(d_histo))
    gpuErrorcheck(hipFree(d_degree))

            return 0;
}
