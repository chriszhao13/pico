#include "hip/hip_runtime.h"
//
// Created by Chris on 2022/11/30.
//
#include "shared/graph.h"
#include "shared/Timer.h"
#include "shared/util.h"
#include "shared/virtual_graph.h"
#include "shared/globals.h"
#include "shared/argument.h"
#include "shared/gpu_error.cuh"

// __device__ int DEVICE_atomicsub = 0;

__global__ void kernel(
        uint partSize,

        bool *sighed,

        unsigned int numParts,

        unsigned int *nodePointer,

        PartPointer *partNodePointer,

        unsigned int *edgeList,

        unsigned int *dist,

        bool *finished,

        bool *plus,

        unsigned int level) {

    unsigned int partId = blockDim.x * blockIdx.x + threadIdx.x;

    if (partId < numParts) {

        unsigned int id = partNodePointer[partId].node;
        unsigned int part = partNodePointer[partId].part;

        if (sighed[partId]) {
            return;
        }

        *finished = false;

        if (dist[id] != level) {
            return;
        }

        *plus = false;
        sighed[partId] = true;

        unsigned int thisPointer = nodePointer[id];
        unsigned int degree = edgeList[thisPointer];

        int numParts;


        if (degree % partSize == 0)

            numParts = degree / partSize;

        else

            numParts = degree / partSize + 1;

        unsigned int end;

        unsigned int ofs = thisPointer + part + 1;

        for (int i = 0; i < partSize; i++) {

            if (part + i * numParts >= degree)
                break;

            end = ofs + i * numParts;

            if (dist[edgeList[end]] > level) {

                if (atomicSub(dist + edgeList[end], 1) == level) {

                    dist[edgeList[end]] = level;
                }
            }
        }

    }

}



int main(int argc, char **argv) {

    argument arguments(argc, argv, false, false);

    Graph graph(arguments.input, false);

    graph.ReadGraph();

    VirtualGraph vGraph(graph, arguments.Part_Size);

    vGraph.MakeUGraph();

    uint Part_Size = arguments.Part_Size;

    uint num_nodes = graph.num_nodes;
    uint num_edges = 2 * graph.num_edges;

    hipSetDevice(arguments.deviceID);
    hipFree(0);

    uint *dist = new uint[num_nodes];

    for (int i = 0; i < num_nodes; i++) {
        dist[i] = vGraph.outDegree[i];
    }
    unsigned int *d_nodePointer;
    unsigned int *d_edgeList;
    unsigned int *d_dist;
    PartPointer *d_partNodePointer;

    bool finished = false;
    bool *d_finished;

    bool plus = true;
    bool *d_plus;

    bool *d_sighed;

    gpuErrorcheck(hipMalloc(&d_nodePointer, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_edgeList, (num_edges + num_nodes) * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_dist, num_nodes * sizeof(unsigned int)));
    gpuErrorcheck(hipMalloc(&d_finished, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_plus, sizeof(bool)));
    gpuErrorcheck(hipMalloc(&d_partNodePointer, vGraph.numParts * sizeof(PartPointer)));
    gpuErrorcheck(hipMalloc(&d_sighed,  vGraph.numParts * sizeof(bool)));
    gpuErrorcheck(
            hipMemcpy(d_nodePointer, vGraph.nodePointer, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_edgeList, vGraph.edgeList, (num_edges + num_nodes) * sizeof(unsigned int),
                             hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_dist, dist, num_nodes * sizeof(unsigned int), hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemcpy(d_partNodePointer, vGraph.partNodePointer, vGraph.numParts * sizeof(PartPointer),
                             hipMemcpyHostToDevice));
    gpuErrorcheck(hipMemset(d_sighed, false, vGraph.numParts * sizeof(bool)));

   

    int itr = 0;
    uint level = 0;

    plus = true;
    Timer t{};
    t.Start();
    while (!finished) {

        itr++;

        if (plus) {
            level++;
        } else {
            plus = true;
        }

        finished = true;

        hipMemcpy(d_finished, &finished, sizeof(bool), hipMemcpyHostToDevice);
        hipMemcpy(d_plus, &plus, sizeof(bool), hipMemcpyHostToDevice);

        kernel<<< vGraph.numParts / 512 + 1, 512 >>>(
                Part_Size,
                d_sighed,
                vGraph.numParts,
                d_nodePointer,
                d_partNodePointer,
                d_edgeList,
                d_dist,
                d_finished,
                d_plus,
                level);

       hipMemcpy(&finished, d_finished, sizeof(bool), hipMemcpyDeviceToHost);
       hipMemcpy(&plus, d_plus, sizeof(bool), hipMemcpyDeviceToHost);
    }

    double runtime = t.Finish();


    cout << "Number of iterations = " << itr << endl;

    cout << "Processing finished in " << runtime << " (ms).\n";

    gpuErrorcheck(hipMemcpy(dist, d_dist, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));

    utility::PrintMaxMin(dist, num_nodes);

    gpuErrorcheck(hipFree(d_nodePointer));
    gpuErrorcheck(hipFree(d_edgeList));
    gpuErrorcheck(hipFree(d_dist));
    gpuErrorcheck(hipFree(d_finished));
    gpuErrorcheck(hipFree(d_partNodePointer));
    gpuErrorcheck(hipFree(d_sighed));

    return 0;

}




